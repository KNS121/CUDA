#include "hip/hip_runtime.h"
﻿#include"MatrixMulCPU.h"
#include <vector>
#include <stdexcept>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>


#define N 4
#define BLOCK_SIZE 32

dim3 kolvo_potokov(32, 32);
dim3 kolvo_blockov(32,32);

using namespace std;
using std::vector;
using namespace std::chrono;


void fillMatrix(vector<vector<int>>& matrix) {
    for (auto& row : matrix) {
        for (auto& element : row) {
            element = rand() % 100;
        }
    }
}

void printMatrix(const vector<vector<int>>& matrix) {
    for (const auto& row : matrix) {
        for (const auto& element : row) {
            std::cout << element << "\t";
        }
        std::cout << std::endl;
    }
}


bool proverka_results(const vector<vector<int>>& Res_CPU, const vector<vector<int>>& Res_GPU, const int n) {


    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Res_GPU[i][j] != Res_CPU[i][j]) {
                return false;
            }
        }
    }
    return true;
}

#define TILE_SIZE 32
#define WARPSIZE 32



#define WARP_SIZE 32



__global__ void MatrixMultplyGPU_WaprIntr(const int* a, const int* b, int* c, int n) {
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int idx = threadIdx.x;
    int idy = threadIdx.y;

    __shared__ int array_a_in_shared[BLOCK_SIZE][BLOCK_SIZE + 1];
    __shared__ int array_b_in_shared[BLOCK_SIZE][BLOCK_SIZE + 1];

    int res = 0;
    float blocks_pokrytie = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    for (int i = 0; i < blocks_pokrytie; ++i) {
        int index_of_A = idx + i * BLOCK_SIZE;
        int index_of_B = i * BLOCK_SIZE + idy;

        // Загрузка A
        if (row < n && index_of_A < n)
            array_a_in_shared[idy][idx] = a[row * n + index_of_A];
        else
            array_a_in_shared[idy][idx] = 0;

        // Загрузка B с транспонированием!
        if (col < n && index_of_B < n)
            array_b_in_shared[idx][idy] = b[index_of_B * n + col];  // idx ↔ idy
        else
            array_b_in_shared[idx][idy] = 0;

        __syncthreads();

        int thread_element_a = array_a_in_shared[idy][idx];
        int thread_element_b = array_b_in_shared[idy][idx];  // Исправленный доступ

        // Корректный сбор данных
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            int a_val = __shfl_sync(0xFFFFFFFF, thread_element_a, k);
            int b_val = __shfl_sync(0xFFFFFFFF, thread_element_b, k);
            res += a_val * b_val;
        }

        __syncthreads();
    }

    if (row < n && col < n)
        c[row * n + col] = res;
}











vector<vector<int>> MatrixMultCUDA(const vector<vector<int>>& A, const vector<vector<int>>& B, const int n) {

    //one dim arrays
    int* one_dim_array_A = new int[n * n];
    int* one_dim_array_B = new int[n * n];
    int* one_dim_array_reuslt = new int[n * n];

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            one_dim_array_A[i * n + j] = A[i][j];
            one_dim_array_B[i * n + j] = B[i][j];
        }
    }


    // go to CUDA
    int* dev_A;
    int* dev_B;
    int* dev_res;

    hipMalloc(&dev_A, n * n * sizeof(int));
    hipMalloc(&dev_B, n * n * sizeof(int));
    hipMalloc(&dev_res, n * n * sizeof(int));

    hipMemcpy(dev_A, one_dim_array_A, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, one_dim_array_B, n * n * sizeof(int), hipMemcpyHostToDevice);

    MatrixMultplyGPU_WaprIntr << < kolvo_blockov, kolvo_potokov >> > (dev_A, dev_B, dev_res, n);

    // obratno
    hipMemcpy(one_dim_array_reuslt, dev_res, n * n * sizeof(int), hipMemcpyDeviceToHost);

    vector<vector<int>> resultMatrix(n, vector<int>(n, 0));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            resultMatrix[i][j] = one_dim_array_reuslt[i * n + j];
        }
    }

    // pochistim

    delete[] one_dim_array_A;
    delete[] one_dim_array_B;
    delete[] one_dim_array_reuslt;

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_res);


    return resultMatrix;
}







int main() {

    vector<vector<int>> A(N, vector<int>(N));
    vector<vector<int>> B(N, vector<int>(N));



    fillMatrix(A);
    fillMatrix(B);

    cout << "Matrix A\n" << endl;
    printMatrix(A);
    cout << "\n" << endl;
    cout << "Matrix B\n" << endl;
    printMatrix(B);

    cout << "\n" << endl;

    auto start_cpu = chrono::high_resolution_clock::now();
    vector<vector<int>> res_from_CPU = MatrixMultiplyCPU(A, B, N);
    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<double> cpu_time = end_cpu - start_cpu;

    auto start_gpu = chrono::high_resolution_clock::now();
    vector<vector<int>> res_from_GPU = MatrixMultCUDA(A, B, N);
    auto end_gpu = chrono::high_resolution_clock::now();
    chrono::duration<double> gpu_time = end_gpu - start_gpu;


    bool check = proverka_results(res_from_CPU, res_from_GPU, N);

    cout << "Razmer matrix : " << N << " \n";

    cout << "GPU time ( Warp intr ) : " << gpu_time.count() << " secund \n";
    cout << "GPU result:\n";
    printMatrix(res_from_GPU);

    cout << "CPU time: " << cpu_time.count() << " secund \n";
    cout << "CPU result:\n";
    printMatrix(res_from_CPU);



    cout << "proverka rezov - " << check << "\n";

    return 0;
}