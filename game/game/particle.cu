#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include "particle.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
using namespace std;


__constant__ float d_world_size = 2.0f;
const float min_distance = 0.05f;
int numParticles = 256;
int winWidth = 1024, winHeight = 1024;
CudaParams cudaParams;
float deltaTime = 0.016f;

// for cuda
__device__ int calcGridHash(float x, float y) {
    int gridX = static_cast<int>((x + 1.0f) * (GRID_SIZE / (2.0f * d_world_size)));
    int gridY = static_cast<int>((y + 1.0f) * (GRID_SIZE / (2.0f * d_world_size)));
    gridX = max(0, min(gridX, GRID_SIZE - 1));
    gridY = max(0, min(gridY, GRID_SIZE - 1));
    return gridY * GRID_SIZE + gridX;
}

__global__ void sortParticles(float* positions_x, float* positions_y, int* cellIndices, int numParticles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    int hash = calcGridHash(positions_x[idx], positions_y[idx]);
    cellIndices[idx] = (hash << 16) | idx;
}

__global__ void setupGrid(int* cellIndices, int* cellStarts, int* cellEnds, int numParticles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    int currentHash = cellIndices[idx] >> 16;

    // start yacheyki
    if (idx == 0 || currentHash != (cellIndices[idx - 1] >> 16)) {
        cellStarts[currentHash] = idx; 
    }

    // end_yacheyki
    if (idx == numParticles - 1 || currentHash != (cellIndices[idx + 1] >> 16)) {
        cellEnds[currentHash] = idx + 1; 
    }
}

__global__ void updateParticles(float* positions_x, float* positions_y,
    float* velocities_x, float* velocities_y,
    int numParticles, float deltaTime) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    // update pos
    positions_x[idx] += velocities_x[idx] * deltaTime;
    positions_y[idx] += velocities_y[idx] * deltaTime;

    // prik-skok ot kraev
    if (fabsf(positions_x[idx]) >= 1.0f) velocities_x[idx] *= -1.0f;
    if (fabsf(positions_y[idx]) >= 1.0f) velocities_y[idx] *= -1.0f;
}

__global__ void processBlockPrikSkok(
    float* positions_x,
    float* positions_y,
    float* velocities_x,
    float* velocities_y,
    int* types,
    int numParticles,
    float min_dist
) {
    // usaem shared
    __shared__ struct {
        float x[BLOCK_SIZE];
        float y[BLOCK_SIZE];
        int type[BLOCK_SIZE];
        float vx[BLOCK_SIZE];
        float vy[BLOCK_SIZE];
    } shared;

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.x;

    if (idx < numParticles) {
        shared.x[tid] = positions_x[idx];
        shared.y[tid] = positions_y[idx];
        shared.type[tid] = types[idx];
        shared.vx[tid] = velocities_x[idx];
        shared.vy[tid] = velocities_y[idx];
    }
    __syncthreads();

    if (idx >= numParticles) return;

    const int type1 = shared.type[tid];
    float delta_vx = 0.0f, delta_vy = 0.0f;
    float delta_x = 0.0f, delta_y = 0.0f;

    
    for (int i = 0; i < BLOCK_SIZE; ++i) {
        const int type2 = shared.type[i];
        if (type1 == type2 || i == tid) continue;

        const float dx = shared.x[tid] - shared.x[i];
        const float dy = shared.y[tid] - shared.y[i];
        const float distSq = dx * dx + dy * dy;

        if (distSq < min_dist * min_dist && distSq > 1e-8f) {
            const float dist = sqrtf(distSq);
            const float nx = dx / dist, ny = dy / dist;
            const float relVel = (shared.vx[tid] - shared.vx[i]) * nx
                + (shared.vy[tid] - shared.vy[i]) * ny;

            if (relVel >= 0) continue;

            const float impulse = -1.0f * relVel;
            delta_vx += impulse * nx;
            delta_vy += impulse * ny;

            const float overlap = 0.5f * (min_dist - dist);
            delta_x += overlap * nx;
            delta_y += overlap * ny;
        }
    }
    //ot greha podalshe
    atomicAdd(&velocities_x[idx], delta_vx);
    atomicAdd(&velocities_y[idx], delta_vy);
    atomicAdd(&positions_x[idx], delta_x);
    atomicAdd(&positions_y[idx], delta_y);
}

// mezhgdu gridami
__global__ void processGridPrikSkok(
    float* positions_x,
    float* positions_y,
    float* velocities_x,
    float* velocities_y,
    int* types,
    int* cellStarts,
    int* cellEnds,
    int* cellIndices,
    int numParticles,
    float min_dist
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    const float p1_x = positions_x[idx], p1_y = positions_y[idx];
    const float v1_x = velocities_x[idx], v1_y = velocities_y[idx];
    const int type1 = types[idx];

    const int gridHash = calcGridHash(p1_x, p1_y);
    const int gridX = gridHash % GRID_SIZE, gridY = gridHash / GRID_SIZE;

    float delta_vx = 0.0f, delta_vy = 0.0f;
    float delta_x = 0.0f, delta_y = 0.0f;

    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            const int neighborX = gridX + dx, neighborY = gridY + dy;
            if (neighborX < 0 || neighborX >= GRID_SIZE) continue;
            if (neighborY < 0 || neighborY >= GRID_SIZE) continue;

            const int neighborHash = neighborY * GRID_SIZE + neighborX;
            const int start = cellStarts[neighborHash], end = cellEnds[neighborHash];
            if (start == -1 || end == -1) continue;

            for (int k = start; k < end; k++) {
                const int idx2 = cellIndices[k] & 0xFFFF;
                if (idx2 <= idx) continue;

                const int type2 = types[idx2];
                if (type1 == type2) continue;

                const float p2_x = positions_x[idx2], p2_y = positions_y[idx2];
                const float dx = p1_x - p2_x, dy = p1_y - p2_y;
                const float distSq = dx * dx + dy * dy;

                if (distSq < min_dist * min_dist && distSq > 1e-8f) {
                    const float dist = sqrtf(distSq);
                    const float nx = dx / dist, ny = dy / dist;
                    const float relVel = (v1_x - velocities_x[idx2]) * nx
                        + (v1_y - velocities_y[idx2]) * ny;

                    if (relVel >= 0) continue;

                    const float impulse = -1.0f * relVel;
                    delta_vx += impulse * nx;
                    delta_vy += impulse * ny;

                    velocities_x[idx2] -= impulse * nx;
                    velocities_y[idx2] -= impulse * ny;

                    const float overlap = 0.5f * (min_dist - dist);
                    delta_x += overlap * nx;
                    delta_y += overlap * ny;
                    positions_x[idx2] -= overlap * nx;
                    positions_y[idx2] -= overlap * ny;
                }
            }
        }
    }

    atomicAdd(&velocities_x[idx], delta_vx);
    atomicAdd(&velocities_y[idx], delta_vy);
    atomicAdd(&positions_x[idx], delta_x);
    atomicAdd(&positions_y[idx], delta_y);
}


__global__ void ClearTexture(hipSurfaceObject_t surface, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        surf2Dwrite(make_uchar4(0, 0, 0, 255), surface, x * sizeof(uchar4), y);
    }
}

__global__ void drawParticles(hipSurfaceObject_t surface,
    float* positions_x, float* positions_y,
    int* types, int numParticles,
    int width, int height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    float x = positions_x[idx];
    float y = positions_y[idx];
    int type = types[idx];

    int px = static_cast<int>((x + 1.0f) * 0.5f * width);
    int py = static_cast<int>((y + 1.0f) * 0.5f * height);

    uchar4 color = (type == PARTICLE_TYPE_A)
        ? make_uchar4(255, 50, 50, 255)
        : make_uchar4(50, 50, 255, 255);


    for (int dy = -10; dy <= 10; ++dy) {
        for (int dx = -10; dx <= 10; ++dx) {
            int x = px + dx;
            int y = py + dy;
            if (x >= 0 && x < width && y >= 0 && y < height) {
                surf2Dwrite(color, surface, x * sizeof(uchar4), y);
            }
        }
    }
}

// dlya lovli oshibok
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void initCuda() {
    checkCudaError(hipMalloc(&cudaParams.positions_x, MAX_PARTICLES * sizeof(float)), "Alloc positions X");
    checkCudaError(hipMalloc(&cudaParams.positions_y, MAX_PARTICLES * sizeof(float)), "Alloc positions Y");
    checkCudaError(hipMalloc(&cudaParams.velocities_x, MAX_PARTICLES * sizeof(float)), "Alloc velocities X");
    checkCudaError(hipMalloc(&cudaParams.velocities_y, MAX_PARTICLES * sizeof(float)), "Alloc velocities Y");
    checkCudaError(hipMalloc(&cudaParams.types, MAX_PARTICLES * sizeof(int)), "Alloc types");
    checkCudaError(hipMalloc(&cudaParams.cellIndices, MAX_PARTICLES * sizeof(int)), "Alloc cell indices");
    checkCudaError(hipMalloc(&cudaParams.cellStarts, GRID_SIZE * GRID_SIZE * sizeof(int)), "Alloc cell starts");
    checkCudaError(hipMalloc(&cudaParams.cellEnds, GRID_SIZE * GRID_SIZE * sizeof(int)), "Alloc cell ends");

}


void initParticles() {
    std::vector<float> pos_x(numParticles);
    std::vector<float> pos_y(numParticles);
    std::vector<float> vel_x(numParticles);
    std::vector<float> vel_y(numParticles);
    std::vector<int> types(numParticles);

    for (int i = 0; i < numParticles; ++i) {
        pos_x[i] = (rand() % 2000) / 1000.0f - 1.0f;
        pos_y[i] = (rand() % 2000) / 1000.0f - 1.0f;
        vel_x[i] = (rand() % 100 - 100) / 500.0f;
        vel_y[i] = (rand() % 100 - 100) / 500.0f;
        types[i] = rand() % 2;
    }

    hipMemcpy(cudaParams.positions_x, pos_x.data(), numParticles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaParams.positions_y, pos_y.data(), numParticles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaParams.velocities_x, vel_x.data(), numParticles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaParams.velocities_y, vel_y.data(), numParticles * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaParams.types, types.data(), numParticles * sizeof(int), hipMemcpyHostToDevice);
}

void display() {
    
    dim3 block(BLOCK_SIZE);
    dim3 grid((numParticles + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // upldate
    updateParticles << <grid, block >> > (
        cudaParams.positions_x,
        cudaParams.positions_y,
        cudaParams.velocities_x,
        cudaParams.velocities_y,
        numParticles,
        deltaTime
        );
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Update particles kernel");

    // sort po grid
    sortParticles << <grid, block >> > (
        cudaParams.positions_x,
        cudaParams.positions_y,
        cudaParams.cellIndices,
        numParticles
        );
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Sort particles kernel");

    // trust
    thrust::sort(
        thrust::device_ptr<int>(cudaParams.cellIndices),
        thrust::device_ptr<int>(cudaParams.cellIndices + numParticles)
    );

    // init grid
    hipMemset(cudaParams.cellStarts, 0xFF, GRID_SIZE * GRID_SIZE * sizeof(int));
    hipMemset(cudaParams.cellEnds, 0xFF, GRID_SIZE * GRID_SIZE * sizeof(int));

    // stroim grid
    dim3 setupBlock(256);
    dim3 gridSetup((numParticles + setupBlock.x - 1) / setupBlock.x);
    setupGrid << <gridSetup, setupBlock >> > (
        cudaParams.cellIndices,
        cudaParams.cellStarts,
        cudaParams.cellEnds,
        numParticles
        );
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Setup grid kernel");

    // Prik-Skok blok
    processBlockPrikSkok << <grid, block >> > (
        cudaParams.positions_x,
        cudaParams.positions_y,
        cudaParams.velocities_x,
        cudaParams.velocities_y,
        cudaParams.types,
        numParticles,
        min_distance
        );
    hipDeviceSynchronize();
    // prik skok grid
    processGridPrikSkok << <grid, block >> > (
        cudaParams.positions_x,
        cudaParams.positions_y,
        cudaParams.velocities_x,
        cudaParams.velocities_y,
        cudaParams.types,
        cudaParams.cellStarts,
        cudaParams.cellEnds,
        cudaParams.cellIndices,
        numParticles,
        min_distance
        );
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Collision processing kernel");

    
    hipGraphicsMapResources(1, &cudaParams.cudaResource, 0);
    hipArray* array;
    hipGraphicsSubResourceGetMappedArray(&array, cudaParams.cudaResource, 0, 0);

    
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;
    hipSurfaceObject_t surface;
    hipCreateSurfaceObject(&surface, &resDesc);

    
    dim3 clearBlock(32, 32);
    dim3 clearGrid(
        (winWidth + clearBlock.x - 1) / clearBlock.x,
        (winHeight + clearBlock.y - 1) / clearBlock.y
    );
    ClearTexture << <clearGrid, clearBlock >> > (surface, winWidth, winHeight);
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Clear texture kernel");

    
    drawParticles << <grid, block >> > (
        surface,
        cudaParams.positions_x,
        cudaParams.positions_y,
        cudaParams.types,
        numParticles,
        winWidth,
        winHeight
        );
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Draw particles kernel");

    
    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &cudaParams.cudaResource, 0);

    
    glClear(GL_COLOR_BUFFER_BIT);
    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, cudaParams.texture);

    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, 1.0f);
    glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, 1.0f);
    glEnd();

    glDisable(GL_TEXTURE_2D);
    glutSwapBuffers();

    

}

int main(int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitWindowSize(winWidth, winHeight);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutCreateWindow("GPU Particle System");

    glewInit();

    
    glGenTextures(1, &cudaParams.texture);
    glBindTexture(GL_TEXTURE_2D, cudaParams.texture);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, winWidth, winHeight, 0,
        GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);

    hipError_t err = hipGraphicsGLRegisterImage(&cudaParams.cudaResource,
        cudaParams.texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
    checkCudaError(err, "Register OpenGL texture");

    initCuda();
    initParticles();

    glutDisplayFunc(display);
    glutIdleFunc([]() { glutPostRedisplay(); });
    glutMainLoop();

    hipFree(cudaParams.positions_x);
    hipFree(cudaParams.positions_y);
    hipFree(cudaParams.velocities_x);
    hipFree(cudaParams.velocities_y);
    hipFree(cudaParams.types);

    hipFree(cudaParams.cellIndices);
    hipFree(cudaParams.cellStarts);
    hipFree(cudaParams.cellEnds);
    hipGraphicsUnregisterResource(cudaParams.cudaResource);
    glDeleteTextures(1, &cudaParams.texture);

    return 0;
}
