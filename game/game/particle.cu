#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include "particle.cuh"
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>
#include <surface_functions.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h>


struct CudaResources {
    hipGraphicsResource* cudaResource = nullptr;
    hipSurfaceObject_t surface = 0;
    hipArray* array = nullptr;

    GLuint texture = 0;
    GLuint fbo = 0;
} cudaRes;

Particle particles[MAX_PARTICLES];
int numParticles = 0;
int winWidth = 720, winHeight = 720;

Particle* d_particles = nullptr;

float deltaTime = 0.016f;


const float min_distance = 0.1f;

__global__ void DrawParticles(hipSurfaceObject_t surface, Particle* particles,
    int numParticles, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    Particle p = particles[idx];
    int px = static_cast<int>((p.x + 1.0f) * 0.5f * width);
    int py = static_cast<int>((p.y + 1.0f) * 0.5f * height);

    uchar4 color = (p.type == PARTICLE_TYPE_A) ? make_uchar4(255, 0, 0, 255) : make_uchar4(0, 0, 255, 255);

    for (int y = -PARTICLE_SIZE / 2; y < PARTICLE_SIZE / 2; ++y) {
        for (int x = -PARTICLE_SIZE / 2; x < PARTICLE_SIZE / 2; ++x) {
            int xpos = px + x;
            int ypos = py + y;
            if (xpos >= 0 && xpos < width && ypos >= 0 && ypos < height) {
                surf2Dwrite(color, surface, xpos * sizeof(uchar4), ypos);
            }
        }
    }
}

__global__ void ClearTexture(hipSurfaceObject_t surface, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        surf2Dwrite(make_uchar4(0, 0, 0, 255), surface, x * sizeof(uchar4), y); // ����-�������
    }
}


__global__ void UpdateParticles(Particle* particles, int numParticles, float deltaTime) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    particles[idx].x += particles[idx].vx * deltaTime;
    particles[idx].y += particles[idx].vy * deltaTime;


    if (particles[idx].x - PARTICLE_RADIUS < -1.0f || particles[idx].x + PARTICLE_RADIUS > 1.0f) particles[idx].vx *= -1;
    if (particles[idx].y - PARTICLE_RADIUS < -1.0f || particles[idx].y + PARTICLE_RADIUS > 1.0f) particles[idx].vy *= -1;
}


__device__ void ottalkivanie_dvuh(Particle& particle_1, Particle& particle_2, float min_distance) {
    float dx = particle_1.x - particle_2.x;
    float dy = particle_1.y - particle_2.y;

    float distance_between_centers = sqrtf(dx * dx + dy * dy);

    if (distance_between_centers < min_distance && distance_between_centers > 0) {
        float n_x = dx / distance_between_centers;
        float n_y = dy / distance_between_centers;

        float overlap = 0.5f * (min_distance - distance_between_centers);

        particle_1.x -= overlap * n_x;
        particle_1.y -= overlap * n_y;
        particle_2.x += overlap * n_x;
        particle_2.y += overlap * n_y;

        float delta_V_norm_x = (particle_1.vx - particle_2.vx);
        float delta_V_norm_y = (particle_1.vy - particle_2.vy);

        float delta_V_norm = n_x * delta_V_norm_x + n_y * delta_V_norm_y;

        if (delta_V_norm > 0) return;

        float imp = -1.0f * delta_V_norm;

        particle_1.vx += imp * n_x;
        particle_1.vy += imp * n_y;
        particle_2.vx -= imp * n_x;
        particle_2.vy -= imp * n_y;
    }
}

__global__ void many_ootalkivaniya(Particle* particles, int numParticles, float min_distance) {
    __shared__ Particle sharedParticles[MAX_PARTICLES];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // shared
    if (idx < numParticles) {
        sharedParticles[idx] = particles[idx];
    }
    __syncthreads();

    // pary
    if (idx < numParticles) {
        for (int j = idx + 1; j < numParticles; ++j) {
            if (sharedParticles[idx].type != sharedParticles[j].type) {
                ottalkivanie_dvuh(sharedParticles[idx], sharedParticles[j], min_distance);
            }
        }
    }
    __syncthreads();

    // update global memry
    if (idx < numParticles) {
        particles[idx] = sharedParticles[idx];
    }
}




void initGL(int argc, char** argv) {
    glutInit(&argc, argv);
    glutInitWindowSize(winWidth, winHeight);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutCreateWindow("CUDA Particle System");


    glewExperimental = GL_TRUE;
    GLenum err = glewInit();
    if (err != GLEW_OK) {
        fprintf(stderr, "GLEW Error: %s\n", glewGetErrorString(err));
        exit(1);
    }

    //TEX
    glGenTextures(1, &cudaRes.texture);
    glBindTexture(GL_TEXTURE_2D, cudaRes.texture);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, winWidth, winHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    // Cuda registraciya
    hipError_t cudaErr = hipGraphicsGLRegisterImage(&cudaRes.cudaResource, cudaRes.texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA Register Error: %s\n", hipGetErrorString(cudaErr));
        exit(1);
    }

    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
}



void initParticles() {
    numParticles = 16;
    for (int i = 0; i < numParticles; ++i) {
        particles[i].x = (rand() % 1000) / 500.0f - 1.0f;
        particles[i].y = (rand() % 1000) / 500.0f - 1.0f;
        particles[i].type = rand() % 2;
        particles[i].vx = particles[i].vy = (rand() % 100 - 100) / 100.0f;
    }

    hipMalloc((void**)&d_particles, MAX_PARTICLES * sizeof(Particle));
    hipMemcpy(d_particles, particles, MAX_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
}

void display() {
    // map
    hipGraphicsMapResources(1, &cudaRes.cudaResource, 0);
    hipGraphicsSubResourceGetMappedArray(&cudaRes.array, cudaRes.cudaResource, 0, 0);

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cudaRes.array;
    hipCreateSurfaceObject(&cudaRes.surface, &resDesc);

    dim3 blockSize(256);
    dim3 gridSize((numParticles + blockSize.x - 1) / blockSize.x);

    // Handle particle repulsion
    many_ootalkivaniya << <gridSize, blockSize >> > (d_particles, numParticles, min_distance);
    hipDeviceSynchronize();

    UpdateParticles << <gridSize, blockSize >> > (d_particles, numParticles, deltaTime);
    hipDeviceSynchronize();

    // chistim
    dim3 clearBlocks(32, 32);
    dim3 clearGrid((winWidth + 31) / 32, (winHeight + 31) / 32);
    ClearTexture << <clearGrid, clearBlocks >> > (cudaRes.surface, winWidth, winHeight);
    hipDeviceSynchronize();

    // risovlaka
    DrawParticles << <gridSize, blockSize >> > (cudaRes.surface, d_particles, numParticles, winWidth, winHeight);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    // osvobodim resursy
    hipDestroySurfaceObject(cudaRes.surface);
    hipGraphicsUnmapResources(1, &cudaRes.cudaResource, 0);

    // draw tex
    glClear(GL_COLOR_BUFFER_BIT);
    glEnable(GL_TEXTURE_2D);
    glBindTexture(GL_TEXTURE_2D, cudaRes.texture);

    glBegin(GL_QUADS);
    glTexCoord2f(0, 1); glVertex2f(-1, -1);
    glTexCoord2f(1, 1); glVertex2f(1, -1);
    glTexCoord2f(1, 0); glVertex2f(1, 1);
    glTexCoord2f(0, 0); glVertex2f(-1, 1);
    glEnd();

    glutSwapBuffers();
}

void timer(int) {
    glutPostRedisplay();
    glutTimerFunc(16, timer, 0);
}

int main(int argc, char** argv) {
    initGL(argc, argv);
    initParticles();

    glutDisplayFunc(display);
    glutTimerFunc(0, timer, 0);
    glutMainLoop();

    // osvobodim resursy
    hipGraphicsUnregisterResource(cudaRes.cudaResource);
    glDeleteTextures(1, &cudaRes.texture);

    return 0;
}